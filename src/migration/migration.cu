#include "hip/hip_runtime.h"
# include "migration.cuh"

void Migration::set_parameters()
{    
    title = "\033[34mReverse Time Migration\033[0;0m";

    set_main_parameters();

    rbc_ratio = std::stof(catch_parameter("mig_rbc_ratio", parameters)); 
    rbc_varVp = std::stof(catch_parameter("mig_rbc_varVp", parameters)); 
    rbc_length = std::stof(catch_parameter("mig_rbc_length", parameters));

    nb = (int)(rbc_length / dh) + 1;

    set_wavelet();
    set_geometry();    
    set_seismograms();
    set_properties();
    set_coordinates();

    input_folder = catch_parameter("mig_input_folder", parameters);
    output_folder = catch_parameter("mig_output_folder", parameters);

    image = new float[nPoints]();
    sumPs = new float[nPoints]();
    partial = new float[volsize]();

    hipMalloc((void**)&(d_Pr), volsize*sizeof(float));
    hipMalloc((void**)&(d_Prold), volsize*sizeof(float));
    hipMalloc((void**)&(d_image), volsize*sizeof(float));
    hipMalloc((void**)&(d_sumPs), volsize*sizeof(float));

    hipMemset(d_image, 0.0f, volsize*sizeof(float));
    hipMemset(d_sumPs, 0.0f, volsize*sizeof(float));
}

void Migration::show_information()
{
    auto clear = system("clear");

    padding = (WIDTH - title.length() + 8) / 2;

    std::string line(WIDTH, '-');

    std::cout << line << "\n";
    std::cout << std::string(padding, ' ') << title << '\n';
    std::cout << line << "\n\n";

    std::cout << "Model dimensions: (z = " << (nz - 1)*dh << 
                                  ", x = " << (nx - 1)*dh <<
                                  ", y = " << (ny - 1)*dh <<") m\n\n";

    std::cout << "Running shot " << srcId + 1 << " of " << geometry->nrel << " in total\n\n";

    std::cout << "Current shot position: (z = " << geometry->zsrc[geometry->sInd[srcId]] << 
                                       ", x = " << geometry->xsrc[geometry->sInd[srcId]] <<
                                       ", y = " << geometry->ysrc[geometry->sInd[srcId]] << ") m\n\n";

    std::cout << line << "\n";
    std::cout << stage_info << std::endl;
    std::cout << line << "\n";                                                                          
}

void Migration::forward_propagation()
{   
    stage_info = "Forward propagation";

    show_information();

    set_random_boundary(d_Vp, rbc_ratio, rbc_varVp);
    
    initialization();
    forward_solver();
}

void Migration::backward_propagation()
{
    stage_info = "Backward propagation";

    show_information();

    initialization();
    set_seismic_source();

    hipMemset(d_Pr, 0.0f, volsize*sizeof(float));
    hipMemset(d_Prold, 0.0f, volsize*sizeof(float));

    for (int tId = 0; tId < nt + tlag; tId++)
    {
        RTM<<<nBlocks, NTHREADS>>>(d_P, d_Pold, d_Pr, d_Prold, d_Vp, d_seismogram, d_image, d_sumPs, d_rIdx, d_rIdy, d_rIdz, geometry->spread, tId, tlag, nxx, nyy, nzz, nt, dh, dt);
    
        std::swap(d_P, d_Pold);
        std::swap(d_Pr, d_Prold);
    }
}

void Migration::set_seismic_source()
{
    std::string data_file = data_folder + "seismogram_nt" + std::to_string(nt) + "_nr" + std::to_string(geometry->spread) + "_" + std::to_string(int(1e6f*dt)) + "us_shot_" + std::to_string(srcId+1) + ".bin";
    import_binary_float(data_file, seismogram, nt*geometry->spread);
    hipMemcpy(d_seismogram, seismogram, nt*geometry->spread*sizeof(float), hipMemcpyHostToDevice);
}

void Migration::export_seismic()
{
    hipMemcpy(partial, d_image, volsize*sizeof(float), hipMemcpyDeviceToHost);
    reduce_boundary(partial, image);

    hipMemcpy(partial, d_sumPs, volsize*sizeof(float), hipMemcpyDeviceToHost);
    reduce_boundary(partial, sumPs);

    # pragma omp parallel for
    for (int index = 0; index < nPoints; index++)
        image[index] = image[index] / sumPs[index];

    # pragma omp parallel for    
    for (int index = 0; index < nPoints; index++)
    {
        int k = (int) (index / (nx*nz));         
        int j = (int) (index - k*nx*nz) / nz;   
        int i = (int) (index - j*nz - k*nx*nz);      

        if((i > 0) && (i < nz-1) && (j > 0) && (j < nx-1) && (k > 0) && (k < ny-1)) 
        {
            float d2I_dx2 = (image[i + (j-1)*nz + k*nx*nz] - 2.0f*image[index] + image[i + (j+1)*nz + k*nx*nz]) / (dh * dh);
            float d2I_dy2 = (image[i + j*nz + (k-1)*nx*nz] - 2.0f*image[index] + image[i + j*nz + (k+1)*nx*nz]) / (dh * dh);
            float d2I_dz2 = (image[(i-1) + j*nz + k*nx*nz] - 2.0f*image[index] + image[(i+1) + j*nz + k*nx*nz]) / (dh * dh);

            sumPs[index] = d2I_dx2 + d2I_dy2 + d2I_dz2;
        }
        else sumPs[index] = 0.0f;
    }

    std::string output_file = output_folder + "RTM_section_" + std::to_string(nz) + "x" + std::to_string(nx) + "x" + std::to_string(ny) + ".bin";
    export_binary_float(output_file, sumPs, nPoints);
}

__global__ void RTM(float * Ps, float * Psold, float * Pr, float * Prold, float * Vp, float * seismogram, float * image, float * sumPs, int * rIdx, int * rIdy, int * rIdz, int spread, int tId, int tlag, int nxx, int nyy, int nzz, int nt, float dh, float dt)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int k = (int) (index / (nxx*nzz));         
    int j = (int) (index - k*nxx*nzz) / nzz;   
    int i = (int) (index - j*nzz - k*nxx*nzz);    

    if ((index == 0) && (tId < nt))
        for (int rId = 0; rId < spread; rId++)
            Pr[rIdz[rId] + rIdx[rId]*nzz + rIdy[rId]*nxx*nzz] += seismogram[(nt-tId-1) + rId*nt] / (dh*dh); 
    
    if((i > 3) && (i < nzz-4) && (j > 3) && (j < nxx-4) && (k > 3) && (k < nyy-4)) 
    {
        float d2Ps_dx2 = (- 9.0f*(Psold[i + (j-4)*nzz + k*nxx*nzz] + Psold[i + (j+4)*nzz + k*nxx*nzz])
                      +   128.0f*(Psold[i + (j-3)*nzz + k*nxx*nzz] + Psold[i + (j+3)*nzz + k*nxx*nzz])
                      -  1008.0f*(Psold[i + (j-2)*nzz + k*nxx*nzz] + Psold[i + (j+2)*nzz + k*nxx*nzz])
                      +  8064.0f*(Psold[i + (j-1)*nzz + k*nxx*nzz] + Psold[i + (j+1)*nzz + k*nxx*nzz])
                      - 14350.0f*(Psold[i + j*nzz + k*nxx*nzz]))/(5040.0f*dh*dh);

        float d2Ps_dy2 = (- 9.0f*(Psold[i + j*nzz + (k-4)*nxx*nzz] + Psold[i + j*nzz + (k+4)*nxx*nzz])
                      +   128.0f*(Psold[i + j*nzz + (k-3)*nxx*nzz] + Psold[i + j*nzz + (k+3)*nxx*nzz])
                      -  1008.0f*(Psold[i + j*nzz + (k-2)*nxx*nzz] + Psold[i + j*nzz + (k+2)*nxx*nzz])
                      +  8064.0f*(Psold[i + j*nzz + (k-1)*nxx*nzz] + Psold[i + j*nzz + (k+1)*nxx*nzz])
                      - 14350.0f*(Psold[i + j*nzz + k*nxx*nzz]))/(5040.0f*dh*dh);

        float d2Ps_dz2 = (- 9.0f*(Psold[(i-4) + j*nzz + k*nxx*nzz] + Psold[(i+4) + j*nzz + k*nxx*nzz])
                      +   128.0f*(Psold[(i-3) + j*nzz + k*nxx*nzz] + Psold[(i+3) + j*nzz + k*nxx*nzz])
                      -  1008.0f*(Psold[(i-2) + j*nzz + k*nxx*nzz] + Psold[(i+2) + j*nzz + k*nxx*nzz])
                      +  8064.0f*(Psold[(i-1) + j*nzz + k*nxx*nzz] + Psold[(i+1) + j*nzz + k*nxx*nzz])
                      - 14350.0f*(Psold[i + j*nzz + k*nxx*nzz]))/(5040.0f*dh*dh);
        
        float d2Pr_dx2 = (- 9.0f*(Pr[i + (j-4)*nzz + k*nxx*nzz] + Pr[i + (j+4)*nzz + k*nxx*nzz])
                      +   128.0f*(Pr[i + (j-3)*nzz + k*nxx*nzz] + Pr[i + (j+3)*nzz + k*nxx*nzz])
                      -  1008.0f*(Pr[i + (j-2)*nzz + k*nxx*nzz] + Pr[i + (j+2)*nzz + k*nxx*nzz])
                      +  8064.0f*(Pr[i + (j-1)*nzz + k*nxx*nzz] + Pr[i + (j+1)*nzz + k*nxx*nzz])
                      - 14350.0f*(Pr[i + j*nzz + k*nxx*nzz]))/(5040.0f*dh*dh);

        float d2Pr_dy2 = (- 9.0f*(Pr[i + j*nzz + (k-4)*nxx*nzz] + Pr[i + j*nzz + (k+4)*nxx*nzz])
                      +   128.0f*(Pr[i + j*nzz + (k-3)*nxx*nzz] + Pr[i + j*nzz + (k+3)*nxx*nzz])
                      -  1008.0f*(Pr[i + j*nzz + (k-2)*nxx*nzz] + Pr[i + j*nzz + (k+2)*nxx*nzz])
                      +  8064.0f*(Pr[i + j*nzz + (k-1)*nxx*nzz] + Pr[i + j*nzz + (k+1)*nxx*nzz])
                      - 14350.0f*(Pr[i + j*nzz + k*nxx*nzz]))/(5040.0f*dh*dh);

        float d2Pr_dz2 = (- 9.0f*(Pr[(i-4) + j*nzz + k*nxx*nzz] + Pr[(i+4) + j*nzz + k*nxx*nzz])
                      +   128.0f*(Pr[(i-3) + j*nzz + k*nxx*nzz] + Pr[(i+3) + j*nzz + k*nxx*nzz])
                      -  1008.0f*(Pr[(i-2) + j*nzz + k*nxx*nzz] + Pr[(i+2) + j*nzz + k*nxx*nzz])
                      +  8064.0f*(Pr[(i-1) + j*nzz + k*nxx*nzz] + Pr[(i+1) + j*nzz + k*nxx*nzz])
                      - 14350.0f*(Pr[i + j*nzz + k*nxx*nzz]))/(5040.0f*dh*dh);
        
        Ps[index] = dt*dt*Vp[index]*Vp[index]*(d2Ps_dx2 + d2Ps_dy2 + d2Ps_dz2) + 2.0f*Psold[index] - Ps[index];    

        Prold[index] = dt*dt*Vp[index]*Vp[index]*(d2Pr_dx2 + d2Pr_dy2 + d2Pr_dz2) + 2.0f*Pr[index] - Prold[index];
    
        sumPs[index] += Ps[index]*Ps[index]; 
        image[index] += Ps[index]*Pr[index];
    }
}